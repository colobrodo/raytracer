
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>

void check_cuda(hipError_t result, char const *const func, const char *const file, int const line) {
    if (result) {
        auto error = hipGetErrorString(result);
        std::cout << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
        file << ":" << line << " '" << func << "' \n";
        std::cout << error << '\n';
        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}